#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdio>
#include <iostream>
#include ""
#include <random>
#include <algorithm>
#include "json.hpp"
#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/transform.h>

#define JSON_DATA_FILE "data/IFF7-4_ValinskisV_L1_dat_3.json"
#define RESULT_FILE_NAME "data/IFF7-4_ValinskisV_3_res.txt"

#define ENTRY_CNT_MAX 250
#define MAX_STRING_LEN 1024

using namespace std;
using json = nlohmann::json;
using namespace thrust;


struct sPerson
{
    char name[MAX_STRING_LEN];
    int streetNum;
    double balance;
};

typedef struct sPerson sPerson;


// Function prototypes
host_vector<sPerson> deserializeJsonFile(std::string fileName, int *count);
//void saveToFile(std::string fileName, Person outArr[], int outCnt);
void generate_random_array(int* array, size_t size);


// struct crumple {
//     __host__ sPerson operator ()(sPerson accumulator, sPerson item) {
//         int dlen, slen;
//         // Get string lenght
//         for(dlen=0; accumulator.name[dlen]!='\0'; ++dlen); 

//         // add chars til null terminator
//         for(slen=0; item.name[slen]!='\0'; ++slen, ++dlen)
//         {
//             accumulator.name[dlen] = item.name[slen];
//         }   
//         accumulator.streetNum = accumulator.streetNum + item.streetNum;
//         accumulator.balance = accumulator.balance + item.balance;
//         return accumulator;
//     }
// };

struct add_int
{
    __device__ int operator ()(int accumulator, int item) {
        return accumulator + item;
    }
};

int main() {   
    int peopleCount;

    cout << "Start L3 Thrust..." << endl;

    cout << "Reading parsing JSON file" << JSON_DATA_FILE << endl;
    host_vector<sPerson> people = deserializeJsonFile(JSON_DATA_FILE, &peopleCount);

    // Empty struct for initial accumulator
    sPerson temp;
    temp.name[0] = '\0';
    temp.streetNum = 0;
    temp.balance = 0.0;

    thrust::device_vector<int> deviceVec(peopleCount);
    printf("Count:%d\n", (int)deviceVec.size());


    // thrust::for_each(deviceVec.begin(), deviceVec.end(), [] (sPerson item) { cout << item.name << " \n";});
    // auto res = reduce(people.begin(), people.end(), temp, ;

    // string tmpname(res.name);
    // int tmpst = res.streetNum;
    // double tmpbal = res.balance;

    // Person resp = Person(tmpname,tmpst, tmpbal);
    // cout << resp.InfoHeader();
    // cout << resp.GetStr();

    // cout << "Saving to file..." << endl;

    // std::ofstream ofs(RESULT_FILE_NAME);
    // ofs << resp.InfoHeader(); // print out table header
    // ofs << resp.GetStr();
    // ofs.close();
    
    return 0;
}

host_vector<sPerson> deserializeJsonFile(std::string fileName, int *count)
{
    // Read json file
    std::ifstream i(JSON_DATA_FILE);
    host_vector<sPerson> tmp(ENTRY_CNT_MAX);

    // Create json object
    json j;
    i >> j;

    // Deserialize json
    *count = 0;
    for (auto &x : j.items())
    {
        sPerson tmpPerson;
        strcpy(tmpPerson.name, x.value()["Name"].get<std::string>().c_str());
        tmpPerson.streetNum = x.value()["StreetNum"].get<int>();
        tmpPerson.balance = x.value()["Balance"].get<double>();
        tmp.push_back(tmpPerson);
        (*count)++;
    }

    return tmp;
}

// Saves people data structure to text file
// void saveToFile(std::string fileName, Person outArr[], int outCnt)
// {
//     std::ofstream ofs(fileName);
//     ofs << outArr[0].InfoHeader(); // print out table header
//     for (auto i = 1; i < outCnt; i++)
//     {
//          ofs << outArr[i].GetStr();
//     }

//     ofs.close();
// }